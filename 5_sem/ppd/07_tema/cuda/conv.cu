#include <cuda_device_runtime_api.h>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <fstream>
#include <algorithm>
#include <iostream>

#include <hip/hip_runtime.h>

struct PaddedMat {
  int row_no;
  int col_no;

  double *buf;
  double *padding_buf;

  __device__ PaddedMat() { }

  __device__ PaddedMat(
    int row_no,
    int col_no,
    double *buf,
    double *padding_buf
  )
  : row_no(row_no)
  , col_no(col_no)
  , buf(buf)
  , padding_buf(padding_buf)
  {
    
  }

  __device__ double at_wrapped_copy(int row, int col) {
    if (row < 0)             row = 0;
    if (row >= this->row_no) row = this->row_no-1;
    if (col < 0)             col = 0;
    if (col >= this->col_no) col = this->col_no-1;

    if (row == 0) {
      return this->padding_buf[(0       * this->col_no) + col];
    } else if (row == this->row_no - 1) {
      return this->padding_buf[(1       * this->col_no) + col];
    } else {
      return this->buf[((row-1) * this->col_no) + col];
    }
  }
};

struct Mat {
  int row_no;
  int col_no;

  double *buf;

  __device__ Mat() {}

  __device__ Mat(
    int row_no,
    int col_no,

    double *buf
  )
  : row_no(row_no)
  , col_no(col_no)
  , buf(buf)
  {
    
  }

  __device__ double at_wrapped_copy(int row, int col) {
    if (row < 0)             row = 0;
    if (row >= this->row_no) row = this->row_no-1;
    if (col < 0)             col = 0;
    if (col >= this->col_no) col = this->col_no-1;

    return this->buf[row*this->col_no + col];
  }

  __device__ double at_copy(int row, int col) {
    return this->buf[row*this->col_no + col];
  }

  __device__ double *at(int row, int col) {
    if (row < 0 || row >= this->row_no || col < 0 || col >= this->col_no) {
      return nullptr;
    }

    return &this->buf[row*this->col_no + col];
  }
};

__global__ void cuda_worker(
  int mat_row_no,
  int mat_col_no,
  double *mat_buf,

  int conv_row_no,
  int conv_col_no,
  double *conv_buf,

  double *padding_buf,
  double *inter_buf
) {
  int p = blockDim.x;
  int thread_i = threadIdx.x;

  // intermediate
  inter_buf = inter_buf + thread_i * 2 * mat_col_no;
  Mat inter = Mat(
    2,
    mat_col_no,
    inter_buf
  );

  int inter_curr = 0;
  
  // mat
  int row_quot = mat_row_no / p;
  int row_rem = mat_row_no % p;

  int row_start = row_quot*(thread_i  ) + std::min((thread_i  ), row_rem);
  int row_end   = row_quot*(thread_i+1) + std::min((thread_i+1), row_rem)-1;

  padding_buf = padding_buf + thread_i * 2 * mat_col_no;
  PaddedMat mat_padded = PaddedMat(
    row_end-row_start+1 + 2,
    mat_col_no,
    mat_buf + row_start * mat_col_no,
    padding_buf
  );
  Mat mat = Mat(
    mat_row_no,
    mat_col_no,
    mat_buf
  );

  // conv
  Mat conv = Mat(
    conv_row_no,
    conv_col_no,
    conv_buf
  );

  int row_i = 1;
  for (; row_i < mat_padded.row_no-1; ++row_i) {
    // write past result into mat
    bool can_write = (row_i >= 3);
    if (can_write) {
      for (int col_i = 0; col_i < mat_col_no; ++col_i) {
        *mat.at(row_start+row_i-1-2, col_i) = *inter.at(inter_curr, col_i);
      }
    }

    // compute convolution
    for (int col_i = 0; col_i < mat_padded.col_no; ++col_i) {
      double result = 0;
      for (int conv_row_i = 0; conv_row_i < conv_row_no; ++conv_row_i) {
        for (int conv_col_i = 0; conv_col_i < conv_col_no; ++conv_col_i) {
          result += mat_padded.at_wrapped_copy(row_i-1 + conv_row_i, col_i-1 + conv_col_i) * conv.at_copy(conv_row_i, conv_col_i);
        }
      }

      *inter.at(inter_curr, col_i) = result;
    }
    
    // update intermediate mat row indices
    inter_curr = (inter_curr+1) % 2;
  }

  // write final intermediate stuff
  if (mat_padded.row_no-2 >= 1) {
    for (int col_i = 0; col_i < mat_col_no; ++col_i) {
      *mat.at(row_start+row_i-1-2, col_i) = *inter.at(inter_curr, col_i);
    }
    inter_curr = (inter_curr+1) % 2;
    row_i++;
  }

  if (mat_padded.row_no-2 >= 2) {
    for (int col_i = 0; col_i < mat_col_no; ++col_i) {
      *mat.at(row_start+row_i-1-2, col_i) = *inter.at(inter_curr, col_i);
    }
    inter_curr = (inter_curr+1) % 2;
    row_i++;
  }
}

int main(int argc, char **argv) {
  // cli
  int p = std::stoi(argv[1]);

  int mat_row_no = std::stoi(argv[2]); // N
  int mat_col_no = std::stoi(argv[3]); // M
  char *mat_path = argv[4];

  int conv_row_no = 3;
  int conv_col_no = 3;
  char *conv_path = argv[5];

  char *dst_path = argv[6];

  char *time_path = argv[7];

  // read mat
  int mat_buf_size = sizeof(double) * mat_row_no * mat_col_no;
  double *mat_buf = (double *) malloc(mat_buf_size);
  {
    std::ifstream fin(mat_path, std::ios::binary);
    fin.read(reinterpret_cast<char*>(mat_buf), mat_buf_size);
    fin.close();
  }

  // read conv
  int conv_buf_size = sizeof(double) * conv_row_no * conv_col_no;
  double *conv_buf = (double *) malloc(conv_buf_size);
  {
    std::ifstream fin(conv_path, std::ios::binary);
    fin.read(reinterpret_cast<char*>(conv_buf), conv_buf_size);
    fin.close();
  }

  // padding buf
  int padding_buf_size = sizeof(double) * p * 2 * mat_col_no;
  double *padding_buf = (double *) malloc(padding_buf_size);
  {
    int row_quot = mat_row_no / p;
    int row_rem = mat_row_no % p;

    int start_row = 0;
    for (int i = 0; i < p; ++i) {
      int buf_offset = i * 2 * mat_col_no;
      
      int len = row_quot;
      if (i < row_rem) {
        len++;
      }
      int end_row = start_row + len - 1;

      {
        int start_row_to_copy = std::max(start_row-1, 0);
        for (int j = 0; j < mat_col_no; ++j) {
          padding_buf[buf_offset + j] = mat_buf[start_row_to_copy * mat_col_no + j];
        }

        int end_row_to_copy = std::min(end_row+1, mat_row_no-1);
        for (int j = 0; j < mat_col_no; ++j) {
          padding_buf[buf_offset + mat_col_no + j] = mat_buf[end_row_to_copy * mat_col_no + j];
        }
      }

      start_row = end_row + 1;
    }
  }
  
  // allocate memory on device
  double *dev_mat_buf     = nullptr; hipMalloc(&dev_mat_buf    , mat_buf_size    );
  double *dev_conv_buf    = nullptr; hipMalloc(&dev_conv_buf   , conv_buf_size   );
  double *dev_padding_buf = nullptr; hipMalloc(&dev_padding_buf, padding_buf_size);
  double *dev_inter_buf   = nullptr; hipMalloc(&dev_inter_buf  , sizeof(double) * p * 2 * mat_col_no);


  // copy everything to device
  hipMemcpy(dev_mat_buf    , mat_buf    , mat_buf_size    , hipMemcpyHostToDevice);
  hipMemcpy(dev_conv_buf   , conv_buf   , conv_buf_size   , hipMemcpyHostToDevice);
  hipMemcpy(dev_padding_buf, padding_buf, padding_buf_size, hipMemcpyHostToDevice);
  
  // free some stuff
  free(conv_buf);
  free(padding_buf);
  
  // start da workas
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  cuda_worker <<< 1, p >>> (
    mat_row_no,
    mat_col_no,
    dev_mat_buf,

    conv_row_no,
    conv_col_no,
    dev_conv_buf,

    dev_padding_buf,
    dev_inter_buf
  );

  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  {
    double elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
    elapsed /= 1'000'000'000;

    std::ofstream fout(time_path);
    fout << elapsed << "\n";
    fout.close();
  }

  // copy over result
  hipMemcpy(mat_buf, dev_mat_buf, mat_buf_size, hipMemcpyDeviceToHost);

  // free cuda stuff
  hipFree(dev_mat_buf);
  hipFree(dev_conv_buf);
  hipFree(dev_padding_buf);
  hipFree(dev_inter_buf);

  // write result to file
  std::ofstream fout(dst_path, std::ios::binary);
  fout.write(reinterpret_cast<char*>(mat_buf), mat_buf_size);
  fout.close();

  // free mat
  free(mat_buf);
  
  return 0;
}
